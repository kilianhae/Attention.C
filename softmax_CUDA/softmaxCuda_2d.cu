#include <iostream>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void softmax2D_kernel(float *d_in, float *d_out, int M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        // Find max value in the row
        float max_val = d_in[row * N];
        for (int i = 1; i < N; ++i) {
            max_val = fmaxf(max_val, d_in[row * N + i]);
        }
        
        // Subtract max value from each element for numerical stability
        float sum_exp = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum_exp += expf(d_in[row * N + i] - max_val);
        }

        // Calculate softmax for each element in the row
        d_out[row * N + col] = expf(d_in[row * N + col] - max_val) / sum_exp;
    }
}

void softmax2D_cpu(float *d_in, float *d_out, int M, int N) {
    for (int row = 0; row < M; ++row) {
        // Find max value in the row
        float max_val = d_in[row * N];
        for (int i = 1; i < N; ++i) {
            max_val = fmaxf(max_val, d_in[row * N + i]);
        }
        
        // Subtract max value from each element for numerical stability
        float sum_exp = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum_exp += expf(d_in[row * N + i] - max_val);
        }

        // Calculate softmax for each element in the row
        for (int col = 0; col < N; ++col) {
            d_out[row * N + col] = expf(d_in[row * N + col] - max_val) / sum_exp;
        }
    }
}

int main(int argc, char *argv[]) {
    // Parse command-line arguments
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <num_rows> <num_cols>" << std::endl;
        return 1;
    }
    const int M = std::stoi(argv[1]);
    const int N = std::stoi(argv[2]);

    // Allocate memory on host
    float *input_host = (float*)malloc(M * N * sizeof(float));
    float *output_host_cpu = (float*)malloc(M * N * sizeof(float));
    float *output_host_gpu = (float*)malloc(M * N * sizeof(float));

    // Initialize input data
    for (int i = 0; i < M * N; i++) {
        input_host[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate memory on device
    float *input_device, *output_device;
    hipMalloc(&input_device, M * N * sizeof(float));
    hipMalloc(&output_device, M * N * sizeof(float));

    // Transfer input data from host to device
    hipMemcpy(input_device, input_host, M * N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block size
    dim3 blockSize(32, 32);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);

    // Create CUDA events for timing GPU execution
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    // Record start event for GPU
    hipEventRecord(start_gpu);

    // Launch kernel
    softmax2D_kernel<<<gridSize, blockSize>>>(input_device, output_device, M, N);
    hipDeviceSynchronize();

    // Record stop event for GPU
    hipEventRecord(stop_gpu);

    // Transfer output data from device to host
    hipMemcpy(output_host_gpu, output_device, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Synchronize GPU

    // Calculate GPU execution time
    float gpu_time;
    hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);

    // Create CUDA events for timing CPU execution
    hipEvent_t start_cpu, stop_cpu;
    hipEventCreate(&start_cpu);
    hipEventCreate(&stop_cpu);

    // Record start event for CPU
    hipEventRecord(start_cpu);

    // Perform softmax on CPU for verification
    softmax2D_cpu(input_host, output_host_cpu, M, N);

    // Record stop event for CPU
    hipEventRecord(stop_cpu);

    // Synchronize CPU
    hipDeviceSynchronize();

    // Calculate CPU execution time
    float cpu_time;
    hipEventElapsedTime(&cpu_time, start_cpu, stop_cpu);

    // Compare CPU and GPU results
    bool passed = true;
    for (int i = 0; i < M * N; i++) {
        if (fabs(output_host_cpu[i] - output_host_gpu[i]) > 1e-5) {
            std::cout << "CPU and GPU results mismatch at index " << i << ": "
                      << output_host_cpu[i] << " != " << output_host_gpu[i] << std::endl;
            passed = false;
            break;
        }
    }

    if (passed) {
        std::cout << "CPU and GPU results match." << std::endl;
    } else {
        std::cout << "CPU and GPU results mismatch." << std::endl;
    }

    std::cout << "GPU Execution Time: " << gpu_time << " ms" << std::endl;
    std::cout << "CPU Execution Time: " << cpu_time << " ms" << std::endl;

    // Free memory
    free(input_host);
    free(output_host_cpu);
    free(output_host_gpu);
    hipFree(input_device);
    hipFree(output_device);

    // Destroy events
    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);
    hipEventDestroy(start_cpu);
    hipEventDestroy(stop_cpu);

    return 0;
}
