#include <iostream>
#include <cmath>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

// Kernel declaration
__global__ void softmax_kernel_naive(float *input, float *output, int rows, int cols) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < rows) {
        float max_val = input[idx * cols];
        for (int i = 1; i < cols; i++) {
            if (input[idx * cols + i] > max_val) {
                max_val = input[idx * cols + i];
            }
        }
        float sum_exp = 0.0f;
        for (int i = 0; i < cols; i++) {
            sum_exp += expf(input[idx * cols + i] - max_val);
        }
        for (int i = 0; i < cols; i++) {
            output[idx * cols + i] = expf(input[idx * cols + i] - max_val) / sum_exp;
        }
    }
}

// CPU implementation of softmax
void softmax_cpu(float *input, float *output, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        float max_val = input[i * cols];
        for (int j = 1; j < cols; j++) {
            if (input[i * cols + j] > max_val) {
                max_val = input[i * cols + j];
            }
        }
        float sum_exp = 0.0f;
        for (int j = 0; j < cols; j++) {
            sum_exp += expf(input[i * cols + j] - max_val);
        }
        for (int j = 0; j < cols; j++) {
            output[i * cols + j] = expf(input[i * cols + j] - max_val) / sum_exp;
        }
    }
}

int main() {
    const int rows = 8192; // Example: large number of rows
    const int cols = 8192;  // Example: large number of columns

    // Allocate memory on host
    float *input_host = (float*)malloc(rows * cols * sizeof(float));
    float *output_host_cpu = (float*)malloc(rows * cols * sizeof(float));
    float *output_host_gpu = (float*)malloc(rows * cols * sizeof(float));

    // Initialize input data
    for (int i = 0; i < rows * cols; i++) {
        input_host[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate memory on device
    float *input_device, *output_device;
    hipMalloc(&input_device, rows * cols * sizeof(float));
    hipMalloc(&output_device, rows * cols * sizeof(float));

    // Transfer input data from host to device
    hipMemcpy(input_device, input_host, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block size
    int blockSize = 256;
    int numBlocks = (rows + blockSize - 1) / blockSize;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event for GPU
    hipEventRecord(start);

    // Launch kernel
    softmax_kernel_naive<<<numBlocks, blockSize>>>(input_device, output_device, rows, cols);
    hipDeviceSynchronize();

    // Record stop event for GPU
    hipEventRecord(stop);

    // Transfer output data from device to host
    hipMemcpy(output_host_gpu, output_device, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate GPU execution time
    float gpu_time;
    hipEventElapsedTime(&gpu_time, start, stop);

    // Record start time for CPU
    hipEventRecord(start);

    // Perform softmax on CPU for verification
    softmax_cpu(input_host, output_host_cpu, rows, cols);

    // Record stop time for CPU
    hipEventRecord(stop);

    // Synchronize event recording
    hipEventSynchronize(stop);

    // Calculate CPU execution time
    float cpu_time;
    hipEventElapsedTime(&cpu_time, start, stop);

    // Compare CPU and GPU results
    bool passed = true;
    for (int i = 0; i < rows * cols; i++) {
        if (fabs(output_host_cpu[i] - output_host_gpu[i]) > 1e-5) {
            std::cout << "CPU and GPU results mismatch at index " << i << ": "
                      << output_host_cpu[i] << " != " << output_host_gpu[i] << std::endl;
            passed = false;
            break;
        }
    }

    if (passed) {
        std::cout << "CPU and GPU results match." << std::endl;
    } else {
        std::cout << "CPU and GPU results mismatch." << std::endl;
    }

    std::cout << "GPU Execution Time: " << gpu_time << " ms" << std::endl;
    std::cout << "CPU Execution Time: " << cpu_time << " ms" << std::endl;

    // Free memory
    free(input_host);
    free(output_host_cpu);
    free(output_host_gpu);
    hipFree(input_device);
    hipFree(output_device);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
