#include "hip/hip_runtime.h"
//#include <torch/types.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <ctime>
#include <chrono>
#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
// #include <hipDNN.h>

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
#define H 8
#define BB 1

//Optimized
const int TILE_DIM_Y = 32;  // Tile dimension for rows
const int TILE_DIM_X = 32;  // Tile dimension for columns// must be 32 for this method
const int BLOCK_SIZE = 32;


#include <iostream>
#include <math.h>
__global__ void softmax_kernel_coalesced_coarsened(float *input, float *output, int rows, int cols, int coarsening_factor) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < rows) {
        float max_val = input[idx * cols];
        for (int i = 1; i < cols; i++) {
            float val = input[idx * cols + i];
            max_val = (val > max_val) ? val : max_val;
        }

        float sum_exp = 0.0f;
        // Thread coarsening: each thread handles multiple elements
        for (int i = 0; i < cols; i += coarsening_factor) {
            float exp_sum = 0.0f;
            // Compute the sum of exponentials for the coarsened group
            for (int j = 0; j < coarsening_factor && i + j < cols; j++) {
                float exp_val = expf(input[idx * cols + i + j] - max_val);
                output[idx * cols + i + j] = exp_val;
                exp_sum += exp_val;
            }
            // Accumulate the sum of exponentials for normalization
            sum_exp += exp_sum;
        }
        // Normalize the softmax values
        for (int i = 0; i < cols; i += coarsening_factor) {
            for (int j = 0; j < coarsening_factor && i + j < cols; j++) {
                output[idx * cols + i + j] /= sum_exp;
            }
        }
    }
}

__global__ void softmax_kernel_naive(float *input, float *output, int rows, int cols) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < rows) {
        float max_val = input[idx * cols];
        for (int i = 1; i < cols; i++) {
            if (input[idx * cols + i] > max_val) {
                max_val = input[idx * cols + i];
            }
        }
        float sum_exp = 0.0f;
        for (int i = 0; i < cols; i++) {
            sum_exp += expf(input[idx * cols + i] - max_val);
        }
        for (int i = 0; i < cols; i++) {
            output[idx * cols + i] = expf(input[idx * cols + i] - max_val) / sum_exp;
        }
    }
}

__global__ void softmax2D_kernel(float *d_in, float *d_out, int M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        // Find max value in the row
        float max_val = d_in[row * N];
        for (int i = 1; i < N; ++i) {
            max_val = fmaxf(max_val, d_in[row * N + i]);
        }
        
        // Subtract max value from each element for numerical stability
        float sum_exp = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum_exp += expf(d_in[row * N + i] - max_val);
        }

        // Calculate softmax for each element in the row
        d_out[row * N + col] = expf(d_in[row * N + col] - max_val) / sum_exp;
    }
}



template <typename T>
__global__ void softmaxKernel2D_rows(const T* input, T* exp_sums, int N, int M) {
    int row = blockIdx.y * TILE_DIM_Y + threadIdx.y;
    int col = blockIdx.x * TILE_DIM_X + threadIdx.x;
    T val = 0;
    // Copy data from global memory to shared memory
    if (row < N && col < M) {
        T max_val = input[row * M];  // Initialize max_val with the first element of the row
        for (int i = 1; i < M; ++i) {
            max_val = max(max_val, input[row * M + i]);
        }
        if (sizeof(T) == 8)
            val = exp(input[row * M + col] - max_val);
        else
            val = expf(input[row * M + col] - max_val);
    }
    // warp shuffle reduction
    // Use XOR mode to perform butterfly reduction
    for (int i = 16; i >= 1; i >>= 1)
        val += __shfl_xor_sync(0xffffffff, val, i, 32);
    // update global value for row
    if ((threadIdx.x == 0) && (row < N)) atomicAdd(exp_sums + row, val);
}

template <typename T>
__global__ void softmaxKernel2D_elementwise(const T* input, const T* exp_sums, T* output, int N, int M) {
    int row = blockIdx.y * TILE_DIM_Y + threadIdx.y;
    int col = blockIdx.x * TILE_DIM_X + threadIdx.x;
    // Compute the softmax values
    if (row < N && col < M) {
        T max_val = input[row * M];  // Initialize max_val with the first element of the row
        for (int i = 1; i < M; ++i) {
            max_val = max(max_val, input[row * M + i]);
        }
        T exp_sum_row = exp_sums[row];
        if (sizeof(T) == 8)
            output[row * M + col] = exp(input[row * M + col] - max_val) / exp_sum_row;
        else
            output[row * M + col] = expf(input[row * M + col] - max_val) / exp_sum_row;
    }
}




__global__ void softmax_kernel_naive_batched(float *input, float *output, int batch_size, int n_head, int seq_len, int head_embd) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < batch_size * n_head) {
        int batch_idx = idx / n_head;
        int head_idx = idx % n_head;

        for (int i = 0; i < seq_len; i++) {
            // Compute max value within the sequence for each batch and head
            float max_val = input[(batch_idx * n_head * seq_len * head_embd) + (head_idx * seq_len * head_embd) + i * head_embd];
            for (int j = 1; j < head_embd; j++) {
                float val = input[(batch_idx * n_head * seq_len * head_embd) + (head_idx * seq_len * head_embd) + i * head_embd + j];
                if (val > max_val) {
                    max_val = val;
                }
            }

            // Compute softmax for each element in the sequence
            float sum_exp = 0.0f;
            for (int j = 0; j < head_embd; j++) {
                float val = input[(batch_idx * n_head * seq_len * head_embd) + (head_idx * seq_len * head_embd) + i * head_embd + j];
                sum_exp += expf(val - max_val);
            }

            for (int j = 0; j < head_embd; j++) {
                float val = input[(batch_idx * n_head * seq_len * head_embd) + (head_idx * seq_len * head_embd) + i * head_embd + j];
                output[(batch_idx * n_head * seq_len * head_embd) + (head_idx * seq_len * head_embd) + i * head_embd + j] = expf(val - max_val) / sum_exp;
            }
        }
    }
}


void softmax_cudnn(float *input, float *output, int num_samples, int num_classes) {
    // Set up cuDNN
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    
    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnCreateTensorDescriptor(&output_desc);
    
    hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_samples, num_classes, 1, 1);
    hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_samples, num_classes, 1, 1);
    
    // Perform softmax operation
    float alpha = 1.0f, beta = 0.0f;
    hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha, input_desc, input, &beta, output_desc, output);
    
    // Clean up
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroy(cudnn);
}

void run_softmax_naive(float* A, float* C, int M, int N){
    //const int seq_len = A.size(2);
    //const int head_embd = A.size(3);
    //const int M = seq_len;
    //const int N = head_embd;
    int threadsPerBlock = 256;
    dim3 blockDim(threadsPerBlock);
    dim3 gridDim(ceil((M + blockDim.x - 1) / (float)blockDim.x), ceil((N + blockDim.y - 1) / (float)blockDim.y));

    // loop over batchsize and head
    for (int i = 0; i < BB; i++) {
        for (int j = 0; j <H); j++) {
            // get the i-th batch and j-th head

		float* Cij = &C[i*H*K*N+j*K*N]
		float* Aij = &A[i*H*K*N+j*K*N]

//            torch::Tensor Aij = A[i][j];
//            torch::Tensor Cij = C[i][j];
            // compute the softmax


            softmax_kernel_naive<<<gridDim, blockDim>>>(Aij, Cij, M, N);

        }
    }

}
void run_softmax_batched_naive(torch::Tensor A, torch::Tensor C){
    auto A_data = A.data_ptr<float>();
    auto C_data = C.data_ptr<float>();
    const int batch_size = A.size(0);
    const int n_head = A.size(1);
    const int seq_len = A.size(2);
    const int head_embd = A.size(3);
    const int M = seq_len;
    const int N = head_embd;

    int threadsPerBlock = 256;
    dim3 threadsperblock(threadsPerBlock);

    dim3 blocksPerGrid ((batch_size * n_head + threadsPerBlock - 1) / threadsPerBlock);


    softmax_kernel_naive_batched<<<blocksPerGrid, threadsperblock>>>(A_data, C_data, batch_size, n_head, seq_len, head_embd);


}
/*
void run_softmax_cuDNN(torch::Tensor A, torch::Tensor C){
    const int seq_len = A.size(2);
    const int head_embd = A.size(3);
    const int M = seq_len;
    const int N = head_embd;
    int threadsPerBlock = 256;
    dim3 blockDim(threadsPerBlock);
    dim3 gridDim(ceil((M + blockDim.x - 1) / (float)blockDim.x), ceil((N + blockDim.y - 1) / (float)blockDim.y));

    // loop over batchsize and head
    for (int i = 0; i < BB; i++) {
        for (int j = 0; j < H; j++) {
            // get the i-th batch and j-th head
            torch::Tensor Aij = A[i][j];
            torch::Tensor Cij = C[i][j];

            // compute the softmax

            softmax_cudnn(Aij.data_ptr<float>(), Cij.data_ptr<float>(), M, N);

            //softmaxKernel2D_rows<<<blocks, threads>>>(Aij.data_ptr<float>(), Cij.data_ptr<float>(), M, N);
            //softmaxKernel2D_elementwise<<<blocks, threads>>>(Aij.data_ptr<float>(), d_sums, Cij.data_ptr<float>(),  M, N);
        }
    }

}
*/

void run_softmax_thread_coarse(torch::Tensor A, torch::Tensor C){
    const int seq_len = A.size(2);
    const int head_embd = A.size(3);
    const int M = seq_len;
    const int N = head_embd;
    int threadsPerBlock = 256;
    dim3 blockDim(threadsPerBlock);
    dim3 gridDim(ceil((M + blockDim.x - 1) / (float)blockDim.x), ceil((N + blockDim.y - 1) / (float)blockDim.y));

    // loop over batchsize and head
    for (int i = 0; i < A.size(0); i++) {
        for (int j = 0; j < A.size(1); j++) {
            // get the i-th batch and j-th head
            torch::Tensor Aij = A[i][j];
            torch::Tensor Cij = C[i][j];

            // compute the softmax

            softmax_kernel_coalesced_coarsened(Aij.data_ptr<float>(), Cij.data_ptr<float>(), M, N, 4);

            //softmaxKernel2D_rows<<<blocks, threads>>>(Aij.data_ptr<float>(), Cij.data_ptr<float>(), M, N);
            //softmaxKernel2D_elementwise<<<blocks, threads>>>(Aij.data_ptr<float>(), d_sums, Cij.data_ptr<float>(),  M, N);
        }
    }

}

void run_softmax_optimized(torch::Tensor A, torch::Tensor C){
    const int seq_len = A.size(2);
    const int head_embd = A.size(3);
    const int M = seq_len;
    const int N = head_embd;
    float *d_sums;
    hipMalloc(&d_sums, M * sizeof(float));
    dim3 threads(TILE_DIM_X, TILE_DIM_Y);
    dim3 blocks((M + TILE_DIM_X - 1) / TILE_DIM_X, (N + TILE_DIM_Y - 1) / TILE_DIM_Y);

    // loop over batchsize and head
    for (int i = 0; i < A.size(0); i++) {
        for (int j = 0; j < A.size(1); j++) {
            // get the i-th batch and j-th head
            torch::Tensor Aij = A[i][j];
            torch::Tensor Cij = C[i][j];

            // compute the softmax
            softmaxKernel2D_rows<<<blocks, threads>>>(Aij.data_ptr<float>(), d_sums, M, N);
            softmaxKernel2D_elementwise<<<blocks, threads>>>(Aij.data_ptr<float>(), d_sums, Cij.data_ptr<float>(),  M, N);
        }
    }
    hipFree(d_sums);
}

/*************************************************************************** Invocations*********************************************************/

/*
torch::Tensor forward(torch::Tensor A) {
    const int batch_size = A.size(0);
    const int n_head = A.size(1);
    const int seq_len = A.size(2);
    const int head_embd = A.size(3);
    const int M = seq_len;
    const int N = head_embd;
    double start, end;
    start = getTimeStamp();
    torch::Tensor C = torch::zeros({batch_size, n_head, M, N}, A.options().device(torch::kCUDA));
    
    run_softmax_optimized(A,C);

    //softmax_kernel_naive<<<gridDim, blockDim>>>(A_data, C_data, M, N, softmax_scale);
    hipDeviceSynchronize();
    //hipFree(d_sums);

    end = getTimeStamp();
    printf("Time taken: %lf\n", (end-start));
    return C;
}


*/

int main(){
int M = 4096; // number of rows in dataset
    int N = 4096; 
    std::vector<float> A(BB * H * N * K,1.0);
    std::vector<float> C(BB * H * N * M,1.0);
    float *d_A, *d_C;

    hipMalloc(&d_A, BB * H * M * N * sizeof(float));
    hipMalloc(&d_C, BB * H * M * N * sizeof(float));
    hipMemset(d_C, 0, N*sizeof(float));

    hipMemcpy(d_A,  A.data(),  BB * H * M * N * sizeof(float), hipMemcpyHostToDevice);
   
    hipDeviceSynchronize();


    double start, end;
    start = getTimeStamp();
    run_softmax_naive(d_A,d_C,M,N);
    
    hipDeviceSynchronize();
    end = getTimeStamp();
    std::cout << "Time taken kernel: " << end - start << std::endl;


    hipMemcpy(C.data(),d_C, BB * H * N * M * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(d_A);
    hipFree(d_C);


    return 0;


}
