#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define TILE_DIM 32
#define BLOCK_ROWS 8
#define BLOCK_DIM 32
// const int NUM_REPS = 100;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code == hipSuccess) return;
    fprintf(stderr,"Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
}

double timeStamp() {
    struct timeval tv; 
    gettimeofday(&tv, NULL);
    return tv.tv_usec / 1000.0 + tv.tv_sec;
}

void displayResults(float *A, float *T, int M, int N){
    // display results
	printf("Matrix A: \n");
	printf("----------\n");
	for (int i = 0; i < M; ++i) {
		for (int j = 0; j < N; ++j) {
			printf("A: %f ", A[i * N + j]);
		}
		printf("\n");
	}

	printf("----------\n");
	printf("Transpose: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			printf("%f ", T[i * M + j]);
		}
		printf("\n");
	}
}

void transposeCPU(float *A, float *T, int M, int N) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            T[j * M + i] = A[i * N + j];
        }
    }
}

void validateResults(float *h_A, float *h_T, int M, int N){
    // Allocate memory for the transpose matrix on CPU
    float *h_T_CPU = (float *)malloc(M * N * sizeof(float));
    // Transpose matrix A on CPU
    transposeCPU(h_A, h_T_CPU, M, N);

    // Validate the results
    int incorrectCount = 0;
    for (int i = 0; i < M * N; ++i) {
        if (abs(h_T_CPU[i] - h_T[i]) > 1e-5) {
            incorrectCount++;
            // Uncomment the next line to print each incorrect element
            // printf("Mismatch at index %d, CPU: %f, GPU: %f\n", i, h_T_CPU[i], h_T[i]);
        }
    }
    
    if (incorrectCount == 0) {
        printf("Validation Passed!\n");
    } else {
        printf("Validation Failed: %d elements incorrect.\n", incorrectCount);
    }

    // Clean up CPU transpose matrix
    free(h_T_CPU);
}

__global__ void transposeNaive(float *d_A, float *d_T, int M, int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// swap elements via transpose
	if (row < M && col < N) {
		d_T[col * M + row] = d_A[row * N + col];
	}
}

__global__ void transposeSharedMem(float *d_A, float *d_T, int M, int N) {
	__shared__ float tile[TILE_DIM][TILE_DIM+1];
	
	unsigned int row = blockIdx.y * TILE_DIM + threadIdx.y;
	unsigned int col = blockIdx.x * TILE_DIM + threadIdx.x;
    unsigned int index_in = row * N + col;
	
    if((row < M) && (col < N) && (index_in < M*N)) {
        tile[threadIdx.y][threadIdx.x] = d_A[index_in];
	}
    
	__syncthreads();
    
	row = blockIdx.y * TILE_DIM + threadIdx.x;
	col = blockIdx.x * TILE_DIM + threadIdx.y;
	if((row < M) && (col < N)) {
        unsigned int index_out = col * M + row;
		d_T[index_out] = tile[threadIdx.x][threadIdx.y];
	}
}

__global__ void copySharedMem(float *d_A, float *d_T, int M, int N)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];
    // block is 8 x 32 so x is 8 and y is 32
  int x = blockIdx.x * TILE_DIM + threadIdx.x*4; // 0,4,8,12,16,20,24,28
  int y = blockIdx.y * TILE_DIM + threadIdx.y; // 0,1,2,3,...,31
    //   printf("%d",blockIdx.y);
    //   printf("%d",blockIdx.x);
  int width = gridDim.x * TILE_DIM;

    if (x>=N || y>=M){return;}
    // load all your elements into shared memory
    for (int j=0; j<4;j+=1){
        tile[threadIdx.y*TILE_DIM+threadIdx.x*4+j]=d_A[y*N+x+j]; //thread 0: 0,1,2,3, thread 1: 4,5,6,7 ... ,28,29,30,31, loading is done with offset
    }
    
    __syncthreads();
  // shared memory now contain an exact copy of the tile. We need to load this back coalesced now

    // calculate the elelements that this thread will load back and to where it will load back
    //idx=(threadIdx.x*BLOCK_ROWS+j)*TILE_DIM
    int idy = threadIdx.y;
    int idx;

    for (int j = 0; j < 4; j += 1){
        idx=(threadIdx.x*4+j); // 
        d_T[blockIdx.x*TILE_DIM*N + blockIdx.y*TILE_DIM + idy*M + idx]=tile[idx*TILE_DIM+idy];
    }}



__global__ void copySharedMem_coalesced(float *d_A, float *d_T, int M, int N)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];
    // block is 8 x 32 so x is 8 and y is 32
    int x = blockIdx.x * TILE_DIM + threadIdx.x; // 0,4,8,12,16,20,24,28
    int y = blockIdx.y * TILE_DIM; // 0,1,2,3,...,31
    //   printf("%d",blockIdx.y);
    //   printf("%d",blockIdx.x);
    int width = gridDim.x * TILE_DIM;

    if (x>=N || y>=M){return;}
    // load all your elements into shared memory
    for (int j=threadIdx.y; j<TILE_DIM+threadIdx.y;j+=BLOCK_ROWS){
        tile[j*TILE_DIM+threadIdx.x]=d_A[(y+j)*N+x]; //thread 0: 0,1,2,3, thread 1: 4,5,6,7 ... ,28,29,30,31, loading is done with offset
    }
    
    __syncthreads();
    // shared memory now contain an exact copy of the tile. We need to load this back coalesced now

    // calculate the elelements that this thread will load back and to where it will load back
    //idx=(threadIdx.x*BLOCK_ROWS+j)*TILE_DIM
    int idy;
    int ix=threadIdx.x*TILE_DIM;
    int xx = blockIdx.x*TILE_DIM*N;
    int yy = blockIdx.y*TILE_DIM;
    for (int j = threadIdx.y; j < TILE_DIM+threadIdx.y; j += BLOCK_ROWS){
        d_T[xx + yy + j*M + threadIdx.x]=tile[ix+j];
    }

}

void run_transpose_cublas(torch::Tensor A, torch::Tensor C) {
    hipError_t cudaStat;  // hipMalloc status
    hipblasStatus_t stat;  // cuBLAS functions status
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);

    const float alpha = 1.0;
    const float beta = 0.0;

    // loop over batchsize and head
    for (int i = 0; i < A.size(0); i++) {
        for (int j = 0; j < A.size(1); j++) {
            // get the i-th batch and j-th head
            torch::Tensor Aij = A[i][j];
            torch::Tensor Cij = C[i][j];

            // perform matrix transposition using hipblasSgeam
            stat = hipblasSgeam(handle,
                               HIPBLAS_OP_T,  // transpose A
                               HIPBLAS_OP_N,  // do not transpose B (NULL)
                               Aij.size(0),  // number of rows of A^T
                               Aij.size(1),  // number of columns of A^T
                               &alpha,
                               Aij.data_ptr<float>(),  // pointer to A
                               Aij.size(0),  // leading dimension of A
                               &beta,
                               NULL,  // B is NULL
                               Aij.size(0),  // set ldb to a valid value
                               Cij.data_ptr<float>(),  // pointer to C
                               Cij.size(1));  // leading dimension of C
        }
    }

    hipblasDestroy(handle);
}


void run_copySharedMem_coalesced(float *d_A, float *d_T, int M, int N){
    dim3 blockDim(8,32); // each thread will process 4 cosnecutive 
	dim3 gridDim((N + 32 - 1)/32, (M + 32 - 1)/32);
    copySharedMem_coalesced<<<gridDim, blockDim>>>(d_A, d_T, M, N);
}


void run_copySharedMem(float *d_A, float *d_T, int M, int N){
    dim3 blockDim(32,8); // each thread will process 4 cosnecutive 
	dim3 gridDim((N + 32 - 1)/32, (M + 32 - 1)/32);
    copySharedMem<<<gridDim, blockDim>>>(d_A, d_T, M, N);
}



torch::Tensor forward(torch::Tensor A) {
    // A and B are 4D tensors in row major format:
    // A = (batchsize, head, M, K)
    double start, end;
    start = timeStamp();
    const int M = A.size(2);
    const int N = A.size(3);

    // Initialize A, Z to host memory, A is MxN and C is NxM. Thus x should be 

    
    torch::Tensor C = torch::zeros({A.size(0), A.size(1), N, M}, A.options().device(torch::kCUDA));
    auto A_data = A.data_ptr<float>();
    auto C_data = C.data_ptr<float>();

	dim3 blockDim(8,32); // each thread will process 4 cosnecutive 
	dim3 gridDim((N + 32 - 1)/32, (M + 32 - 1)/32);
    // dim3 blockDim(BLOCK_DIM, BLOCK_DIM); // each thread will process 4 cosnecutive 
	// dim3 gridDim((N + BLOCK_DIM - 1)/BLOCK_DIM, (M + BLOCK_DIM - 1)/BLOCK_DIM);

   
    //transposeNaive<<<gridDim, blockDim>>>(A_data, C_data, M, N);
    //transposeSharedMem<<<gridDim, blockDim>>>(A_data, C_data, M, N);
    copySharedMem_coalesced<<<gridDim, blockDim>>>(A_data, C_data, M, N);
    //copySharedMem<<<gridDim, blockDim>>>(A_data, C_data, M, N);
    // run_transpose_cublas(A, C);
    hipDeviceSynchronize();
    end = timeStamp();

    printf("GPU execution time: %.4f milliseconds\n", (end-start));

	return C;
}


// % nvcc -arch sm_89 transpose.cu -o transpose
// % transpose