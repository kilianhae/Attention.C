
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hipblas.h>

#define TILE_DIM 32
#define BLOCK_ROWS 8
#define BLOCK_DIM 32

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code == hipSuccess) return;
    fprintf(stderr,"Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
}

double timeStamp() {
    struct timeval tv; 
    gettimeofday(&tv, NULL);
    return tv.tv_usec / 1000.0 + tv.tv_sec;
}

void displayResults(float *A, float *T, int M, int N, int fromIdx, int toIdx){
    // display results
	printf("Matrix A: \n");
	printf("----------\n");
	for (int i = 0; i < M; ++i) {
        if (i >= fromIdx && i < toIdx) {
            for (int j = 0; j < N; ++j) {
                if (j >= fromIdx && j < toIdx) {
                    printf("A: %.2f ", A[i * N + j]);
                } else {
                    continue;
                }
            }
        } else {
            continue;
        }
		printf("\n");
	}

	printf("----------\n");
	printf("Transpose: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
        if (i >= fromIdx && i < toIdx) {
            for (int j = 0; j < M; ++j) {
                if (j >= fromIdx && j < toIdx) {
                    printf("%.2f ", T[i * M + j]);
                } else {
                    continue;
                }
            }
        } else {
            continue;
        }
		printf("\n");
	}
}

void transposeCPU(float *A, float *T, int M, int N) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            T[j * M + i] = A[i * N + j];
        }
    }
}

void validateResults(float *h_A, float *h_T, int M, int N){
    // Allocate memory for the transpose matrix on CPU
    float *h_T_CPU = (float *)malloc(M * N * sizeof(float));
    // Transpose matrix A on CPU
    transposeCPU(h_A, h_T_CPU, M, N);

    // Validate the results
    int incorrectCount = 0;
    for (int i = 0; i < M * N; ++i) {
        if (abs(h_T_CPU[i] - h_T[i]) > 1e-5) {
            incorrectCount++;
            // Uncomment the next line to print each incorrect element
            // printf("Mismatch at index %d, CPU: %f, GPU: %f\n", i, h_T_CPU[i], h_T[i]);
        }
    }
    
    if (incorrectCount == 0) {
        printf("Validation Passed!\n");
    } else {
        printf("Validation Failed: %d elements incorrect.\n", incorrectCount);
    }

    // Clean up CPU transpose matrix
    free(h_T_CPU);
}

__global__ void transposeNaive(float *d_A, float *d_T, int M, int N) {
	int row = blockIdx.y * BLOCK_DIM + threadIdx.y;
	int col = blockIdx.x * BLOCK_DIM + threadIdx.x;

	if (row < M && col < N) {
		d_T[col * M + row] = d_A[row * N + col];
	}
}

__global__ void transposeSharedMem(float *d_A, float *d_T, int M, int N) {
	__shared__ float tile[TILE_DIM][TILE_DIM+1];
	
	unsigned int row = blockIdx.y * TILE_DIM + threadIdx.y;
	unsigned int col = blockIdx.x * TILE_DIM + threadIdx.x;
    unsigned int index_in = row * N + col;
	
    if((row < M) && (col < N) && (index_in < M*N)) {
        tile[threadIdx.y][threadIdx.x] = d_A[index_in];
	}
    
	__syncthreads();
    
	row = blockIdx.y * TILE_DIM + threadIdx.x;
	col = blockIdx.x * TILE_DIM + threadIdx.y;
	if((row < M) && (col < N)) {
        unsigned int index_out = col * M + row;
		d_T[index_out] = tile[threadIdx.x][threadIdx.y];
	}
}




void run_transpose_cublas(float *A, float *C, int M,int N) {
    hipError_t cudaStat;  // cudaMalloc status
    hipblasStatus_t stat;  // cuBLAS functions status
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);

    const float alpha = 1.0;
    const float beta = 0.0;

    // loop over batchsize and head
    for (int i = 0; i < 1; i++) {
        for (int j = 0; j < 1; j++) {
            // get the i-th batch and j-th head
            float* Aij = A;
            float* Cij = C;

            // perform matrix transposition using cublasSgeam
            stat = hipblasSgeam(handle,
                               HIPBLAS_OP_T,  // transpose A
                               HIPBLAS_OP_N,  // do not transpose B (NULL)
                               N,  // number of rows of A^T
                               M,  // number of columns of A^T
                               &alpha,
                               Aij,  // pointer to A
                               N,  // leading dimension of A
                               &beta,
                               NULL,  // B is NULL
                               N,  // set ldb to a valid value
                               Cij,  // pointer to C
                               N);  // leading dimension of C
        }
    }
    hipblasDestroy(handle);
}




int main(int argc, char *argv[]) {
    // Set matrix size
    // int M = atoi(argv[1]);
    // int N = atoi(argv[2]);
    int M = 4096;
    int N = 64;
    if (M <= 0 || N <= 0) return 0;
    size_t bytes = M * N * sizeof(float);

	float *h_A, *h_T;
	float *d_A, *d_T;

	// allocate host memory
    gpuErrchk(hipHostAlloc((void **)&h_A, bytes, hipHostMallocMapped));
    gpuErrchk(hipHostAlloc((void **)&h_T, bytes, hipHostMallocMapped));

    
	// initialize data
	for (int i = 0; i < M * N; ++i) {
        h_A[i] = (float)(rand() % 10 + 1);
	}
    

    // allocate device memory
    gpuErrchk(hipMalloc(&d_A, bytes));
    gpuErrchk(hipMalloc(&d_T, bytes));
    

	// copy host data to device
    
	gpuErrchk(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));

    
	// launch kernel instance
	dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
	dim3 gridDim((N + blockDim.x - 1)/blockDim.x, (M + blockDim.y - 1)/blockDim.y);

    double start_total_GPU = timeStamp();
	//transposeNaive<<<gridDim, blockDim>>>(d_A, d_T, M, N);
	//transposeSharedMem<<<gridDim, blockDim>>>(d_A, d_T, M, N);
    run_transpose_cublas(d_A, d_T, M, N);
    
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    double end_total_GPU = timeStamp();
    float total_GPU_time = end_total_GPU - start_total_GPU;

	// copy result back to host
	gpuErrchk(hipMemcpy(h_T, d_T, bytes, hipMemcpyDeviceToHost));


    printf("GPU execution time: %.4f milliseconds\n", total_GPU_time);

  	// display results
    // displayResults(h_A, h_T, M, N, 30, 40);
    validateResults(h_A, h_T, M, N);

	// clean up data
    gpuErrchk(hipHostFree(h_A));
    gpuErrchk(hipHostFree(h_T));
    gpuErrchk(hipFree(d_A)); 
    gpuErrchk(hipFree(d_T));
    gpuErrchk(hipDeviceReset());

	return 0;
}

// $ nvcc -arch sm_89 transpose_eunjin.cu -o transpose_eunjin
// $ ./transpose_eunjin 5 4

// $ nvcc -arch sm_89 transpose_cuda/transpose_eunjin.cu -o transpose_cuda/transpose_eunjin
// $ transpose_cuda/transpose_eunjin 5 4